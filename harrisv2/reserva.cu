#include "defines.hpp"
void reserva_memoria(parametros &param, int ancho, int alto){
	printf("va a reservar\n");
	//param = (parametros *) malloc(sizeof(parametros));

	param.numBlocks=0;
	param.numThreads=0;

	int nPixels = ancho*alto;
	getNumBlocksAndThreads(6, nPixels, 32, 64, param.numBlocks, param.numThreads);
	param.smemSize = (param.numThreads <= 32) ? 2 * param.numThreads * sizeof(int) : param.numThreads *sizeof(int);




	hipMalloc(&param.d_odata,param.numBlocks*sizeof(int));
	hipMalloc(&param.d_p, nPixels*sizeof(int));
	hipMalloc(&param.d_q, nPixels*sizeof(int));
	hipMalloc(&param.d_pq, nPixels*sizeof(int));
	//hipMalloc(&param.d_pixHist, 1024*sizeof(unsigned int));
	hipMalloc(&param.d_nCandidates,sizeof(int)*2);
	hipMalloc(&param.d_max, 2*sizeof(int));
	hipMalloc(&param.d_input, nPixels*sizeof(unsigned char));
	hipMalloc(&param.d_R, nPixels*sizeof(int));


	/*memset(&param.resDesc, 0, sizeof(param.resDesc));
	param.resDesc.resType = hipResourceTypeLinear;
	param.resDesc.res.linear.devPtr = param.d_R;
	param.resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
	param.resDesc.res.linear.desc.x = 32; // bits per channel
	param.resDesc.res.linear.sizeInBytes = ancho*alto*sizeof(int);*/

	memset(&param.resDesc, 0, sizeof(param.resDesc));
	param.resDesc.resType = hipResourceTypePitch2D;
	param.resDesc.res.pitch2D.devPtr = param.d_R;
	param.resDesc.res.pitch2D.desc.f = hipChannelFormatKindUnsigned;
	param.resDesc.res.pitch2D.desc.x = 32; // bits per channel
	param.resDesc.res.pitch2D.width = ancho;
	param.resDesc.res.pitch2D.height = alto;
	param.resDesc.res.pitch2D.pitchInBytes = ancho*alto*sizeof(int);

	memset(&param.texDesc, 0, sizeof(param.texDesc));
	param.texDesc.readMode = hipReadModeElementType;

	hipCreateTextureObject(&param.texture_array, &param.resDesc, &param.texDesc, NULL);

	hipMalloc(&param.d_pCandidateOffsets,nPixels*sizeof(int));
	//hipMalloc(&param.d_pCovImage, nPixels*sizeof(unsigned char));
	printf("termina de reservar\n");
	return;
}

void libera_memoria(parametros &param){
	hipFree(param.d_odata);
	hipFree(param.d_p);
	hipFree(param.d_q);
	hipFree(param.d_pq);
	hipFree(param.d_nCandidates);
	hipFree(param.d_max);
	hipFree(param.d_input);
	hipFree(param.d_R);
	hipFree(param.d_pCandidateOffsets);
	hipDestroyTextureObject(param.texture_array);
}
