#include "defines.hpp"
void reserva_memoria(parametros &param, int ancho, int alto){
	printf("va a reservar\n");
	//param = (parametros *) malloc(sizeof(parametros));

	param.numBlocks=0;
	param.numThreads=0;

	int nPixels = ancho*alto;
	getNumBlocksAndThreads(6, nPixels, 32, 64, param.numBlocks, param.numThreads);
	param.smemSize = (param.numThreads <= 32) ? 2 * param.numThreads * sizeof(int) : param.numThreads *sizeof(int);

	hipMalloc(&param.d_odata,param.numBlocks*sizeof(int));
	hipMalloc(&param.d_p, nPixels*sizeof(int));
	hipMalloc(&param.d_q, nPixels*sizeof(int));
	hipMalloc(&param.d_pq, nPixels*sizeof(int));
	//hipMalloc(&param.d_pixHist, 1024*sizeof(unsigned int));
	hipMalloc(&param.d_nCandidates,sizeof(int)*2);
	hipMalloc(&param.d_max, 2*sizeof(int));
	hipMalloc(&param.d_input, nPixels*sizeof(unsigned char));
	hipMalloc(&param.d_R, nPixels*sizeof(int));
	hipMalloc(&param.d_pCandidateOffsets,nPixels*sizeof(int));
	//hipMalloc(&param.d_pCovImage, nPixels*sizeof(unsigned char));
	printf("termina de reservar\n");
	return;
}

void libera_memoria(parametros &param){
	hipFree(param.d_odata);
	hipFree(param.d_p);
	hipFree(param.d_q);
	hipFree(param.d_pq);
	hipFree(param.d_nCandidates);
	hipFree(param.d_max);
	hipFree(param.d_input);
	hipFree(param.d_R);
	hipFree(param.d_pCandidateOffsets);
}
