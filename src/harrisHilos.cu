#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : harris_v1.0.cu
 Author      : Julio Gonzalez Carranza
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include "defines.hpp"
#include <thread>
#include <CImg.h>
#include <mutex>
#define __TIEMPO__KERNELS__ true
#define __TIEMPO__TOTAL__ true
#define __GRADIENTE__ 3
#define __HISTOGRAMA__ CPU
#define __UMBRAL__ CPU
#define __HARRIS__ 4
#define __MAXIMO__NO__
//#define __MAXIMO__CPU__
//#define __GOODPIXELS__CPU__
#define __GOODPIXELS__GPU__
#define __SORT__ GPU

#define DEMO_IMAGE	"/home/julio/universidad/proyecto/proyecto/HarrisCornerRAwareEval/corridor/0001.bmp"
#define SRC_IMAGE		"/home/julio/universidad/proyecto/proyecto/HarrisCornerRAwareEval/corridor/"

using namespace std;
using namespace cimg_library;
void GenFileListSorted(char *dirName, char **fileList, int *count)
{
	struct dirent **namelist;
	int n, fileCnt = 0;

	n = scandir(dirName, &namelist, 0, alphasort);
	if (n < 0)
	{
		//printf(" [SMD][ER]Unable to open directory \n");
		*count = 0;
	}
	else
	{
		int iter = 0;

		while (iter < n)
		{
			if((strcmp(namelist[iter]->d_name, ".") != 0) && (strcmp(namelist[iter]->d_name, "..") != 0))
			{
				// process only .bmp files
				if(strstr(namelist[iter]->d_name, ".bmp"))
				{
					fileList[fileCnt] = (char *)malloc(strlen(dirName) + strlen(namelist[iter]->d_name) + 2);
					if(!fileList[fileCnt])
					{
						//printf(" ERROR: malloc failed ! \n");
						return;
					}
		  			strcpy(fileList[fileCnt], dirName);
		  			strcat(fileList[fileCnt], "/");
		  			strcat(fileList[fileCnt], namelist[iter]->d_name);
					printf(" [SMD][OK]File name: %s \n", fileList[fileCnt]);
					free(namelist[iter]);
					fileCnt++;
				}
			}
			iter++;
		}
		*count = fileCnt;
	}
}

void ReadBMP(char* filename, unsigned char* grayData, int &ancho, int &alto)
{
/*    int i;
    FILE* f = fopen(filename, "rb");

    if(f == NULL)
        throw "Argument Exception";

    unsigned char info[54];
    fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

    // extract image height and width from header
    int width = *(int*)&info[18];
    int height = *(int*)&info[22];
*/
    /*cout << endl;
    cout << "  Name: " << filename << endl;
    cout << " Width: " << width << endl;
    cout << "Height: " << height << endl;*/
/*    ancho = width;
    alto = height;
    int row_padded = (width*3 + 3) & (~3);
    unsigned char* data = new unsigned char[row_padded];

    unsigned char tmp;

    for(int i = 0; i < height; i++)
    {
        fread(data, sizeof(unsigned char), row_padded, f);
        for(int j = 0; j < width*3; j += 3)
        {
            // Convert (B, G, R) to (R, G, B)
            tmp = data[j];
            data[j] = data[j+2];
            data[j+2] = tmp;
            //cout << "[" << i*width+j/3 << "]= " << (data[j]+data[j+1]+data[j+2])/3 << endl;

            //funciona
            //grayData[i*width+j/3] = (int)(data[j]+data[j+1]+data[j+2])/3;
            //prueba
            grayData[i*width+j/3]=(data[j]+(data[j+1]<<1)+data[j+2]+2)>>2;
            //funciona
            //grayData[i*width+j/3] = int(data[j]*0.299+data[j+1]*0.587+data[j+2]*0.114);

            //cout << "[" << i*width+j/3 << "]" << endl;
            //cout << "R: "<< (int)data[j] << " G: " << (int)data[j+1]<< " B: " << (int)data[j+2]<< endl;
            //cout << "data["<< i << "," << j/3 << "]= " << (unsigned int)grayData[i*width+j/3] << endl;
            //printf("data: %d\n",grayData[i*width+j/3]);
        }
    }
    free(data);

    fclose(f);
*/
	unsigned char tmp;
	CImg<unsigned char> img(filename);
	alto=img._height;
	ancho=img._width;
	for(int i=0;i<alto;i++){
		for(int j=0;j<ancho;j++){
			tmp=img(j,i,0,0);
			img(j,i,0,0)=img(j,i,0,2);
			img(j,i,0,2)=tmp;
			grayData[i*ancho+j]=(img(j,i,0,0)+img(j,i,0,1)+img(j,i,0,2))/3;
		}
	}
}

int cornerDetector(unsigned char* grayImage, vector<punto> &puntosDeInteres, const int ancho, const int alto, parametros &param, tiempos t_kernel){

	const unsigned char *input = grayImage;
	unsigned int *pixHist;
	const int nPixels = ancho*alto;
	const int width = ancho;
	const int height = alto;
	const int stop = nPixels - width - 1;

	#if __TIEMPO__KERNELS__
		float milliseconds = 0;
		hipEvent_t e_start, e_stop;
		hipEventCreate(&e_start);
		hipEventCreate(&e_stop);
	#endif

	const int diff = HARRIS_WINDOW_SIZE + 1;
	int pOutputImage[width*height];

	int despl = (HARRIS_WINDOW_SIZE / 2) * (width + 1);
	int *R = pOutputImage + (HARRIS_WINDOW_SIZE / 2) * (width + 1);

	int max[1];
	int maximum=0;
	max[0]=0;

	hipMemcpy(param.d_max,max,1*sizeof(int),hipMemcpyHostToDevice);
	/**
	 * Declaracion de los vectores p, q y pq para accesos coalescentes a memoria
	 * de la gpu
	 *
	 */

	hipMemcpy(param.d_input,input,nPixels*sizeof(unsigned char),hipMemcpyHostToDevice);

	/**
	 * Declaracion de dimensiones de grid y tamaño de bloques
	 */
	dim3 dimGrid(width/dimOfBlock,height/dimOfBlock);//numero de tiles
   	dim3 dimBlock(dimOfBlock,dimOfBlock);//tamanio de los tiles

	#if __TIEMPO__KERNELS__
   		hipEventRecord(e_start);
	#endif

	GradientCalc<<<dimGrid,dimBlock>>>(width, height, param.d_input,  stop, param.d_p, param.d_q, param.d_pq, param.d_R,param.d_max);

	#if __TIEMPO__KERNELS__
		hipEventRecord(e_stop);
		hipEventSynchronize(e_stop);
		milliseconds=0;
		hipEventElapsedTime(&milliseconds, e_start, e_stop);
		t_kernel.gradiente+=milliseconds;
		printf("calculo gradiente: %f\n",milliseconds);
	#endif
	hipMemcpy(max,param.d_max,1*sizeof(int),hipMemcpyDeviceToHost);

	free(pixHist);


	//////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////


	//////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////

	#if __TIEMPO__KERNELS__
	hipEventRecord(e_start);
	#endif

	#if __HARRIS__ == 3
	harrisResponseFunction<<<dimGrid,dimBlock>>>(diff, width, height, param.d_R, param.d_p, param.d_q, param.d_pq, 1023,despl);
	#endif

	#if __TIEMPO__KERNELS__
	hipEventRecord(e_stop);
	hipEventSynchronize(e_stop);
	milliseconds=0;
	hipEventElapsedTime(&milliseconds, e_start, e_stop);
	printf("calculo harris: %f\n",milliseconds);
	#endif

	//#ifdef __MAXIMO__CPU__
	hipMemcpy(R,param.d_R+ (HARRIS_WINDOW_SIZE / 2) * (width + 1),(nPixels - (HARRIS_WINDOW_SIZE / 2) * (width + 1))*sizeof(int),hipMemcpyDeviceToHost);
	//#endif
;

	int *data = pOutputImage;
	int *d_data=param.d_R;

	#if __TIEMPO__KERNELS__
	hipEventRecord(e_start);
	#endif
	// determine maximum value

	#ifdef __MAXIMO__GPU__

	int h_odata[param.numBlocks];

	dim3 adimBlock(param.numThreads, 1);
    	dim3 adimGrid(param.numBlocks, 1);

	reduce<<< adimGrid, adimBlock, param.smemSize >>>(d_data, param.d_odata,(unsigned int)nPixels,param.numThreads);
	hipMemcpy(h_odata,param.d_odata,param.numBlocks*sizeof(int),hipMemcpyDeviceToHost);


	for (int i=0; i<param.numBlocks; i++)
            {
                max[1]=maximum=MAX(h_odata[i],maximum );

            }
	#endif

	#ifdef __MAXIMO__CPU__
	maximum = 0;
	for (int i = 0; i < nPixels; ++i)
	{
		if (data[i] > maximum)
			maximum = data[i];

	}
	#endif

	#if __TIEMPO__KERNELS__
		hipEventRecord(e_stop);
		hipEventSynchronize(e_stop);
		milliseconds=0;
		hipEventElapsedTime(&milliseconds, e_start, e_stop);
		printf("maximo: %f\n",milliseconds);
	#endif

	max[0] = int(max[0] * 0.005f);// + 0.5f
	int *pCandidateOffsets = new int[nPixels];
	int *nCandidates = new int[1];
	nCandidates[0]=0;
	// only accept good pixels

	int __GOODPIXELS__ = 1;

	if(__GOODPIXELS__==0){
		for (int i = 0; i < nPixels; i++)
		{
			if (data[i] >= max[0])
				pCandidateOffsets[nCandidates[0]++] = i;
		}
	}

	else if(__GOODPIXELS__==1){
		int *d_aux;

		hipMemcpy(param.d_nCandidates,nCandidates,sizeof(int)*2,hipMemcpyHostToDevice);
		#if __TIEMPO__KERNELS__
		hipEventRecord(e_start);
		#endif
		goodPixels<<<dimGrid,dimBlock>>>(d_data,param.d_pCandidateOffsets, d_aux,param.d_nCandidates,width,height,max[0]);
		#if __TIEMPO__KERNELS__
		hipEventRecord(e_stop);
		hipEventSynchronize(e_stop);
		milliseconds=0;
		hipEventElapsedTime(&milliseconds, e_start, e_stop);
		t_kernel.goodPixels+=milliseconds;
		printf("good pixels: %f\n",milliseconds);
		#endif
		hipMemcpy(nCandidates,param.d_nCandidates,sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(pCandidateOffsets,param.d_pCandidateOffsets,sizeof(int)*nCandidates[0],hipMemcpyDeviceToHost);


		}

	#if __TIEMPO__KERNELS__
		hipEventRecord(e_start);
	#endif
	#if __SORT__ == CPU
	QuicksortInverse(pCandidateOffsets, data, 0, nCandidates[0] - 1);
	//QuicksortInverse(pCandidateOffsets2, data, 0, nCandidates2[0] - 1);
	/*sort(pCandidateOffsets);
	sort();*/
	#endif
	#if __TIEMPO__KERNELS__
		hipEventRecord(e_stop);
		hipEventSynchronize(e_stop);
		milliseconds=0;
		hipEventElapsedTime(&milliseconds, e_start, e_stop);
		t_kernel.qSort+=milliseconds;
		printf("quicksort: %f\n",milliseconds);
	#endif

	//printf(" %f\t",milliseconds);
	float fMinDistance = 5.0f;
	const int nMinDistance = int(fMinDistance );//+ 0.5f
	#if __TIEMPO__KERNELS__
		hipEventRecord(e_start);
	#endif
	unsigned char image[nPixels];
	for (int i=0;i<nPixels;i++) image[i]=0;
	int nInterestPoints = 0;
	const int nMaxPoints=700;
	for (int i = 0; i < nCandidates[0] && nInterestPoints < nMaxPoints; i++)
	{
		const int offset = pCandidateOffsets[i];

		const int x = offset % width;
		const int y = offset / width;

		bool bTake = true;

		const int minx = x - nMinDistance < 0 ? 0 : x - nMinDistance;
		const int miny = y - nMinDistance < 0 ? 0 : y - nMinDistance;
		const int maxx = x + nMinDistance >= width ? width - 1 : x + nMinDistance;
		const int maxy = y + nMinDistance >= height ? height - 1 : y + nMinDistance;
		const int diff = width - (maxx - minx + 1);

		for (int l = miny, offset2 = miny * width + minx; l <= maxy; l++, offset2 += diff)
			for (int k = minx; k <= maxx; k++, offset2++)
				if (image[l * width + k]==1)
				{
					bTake = false;
					break;
				}

		if (bTake)
		{
			// store  point
			//cout << "guarda:" << x << "," << y << endl;
			puntosDeInteres[nInterestPoints].x = float(x);
			puntosDeInteres[nInterestPoints].y = float(y);
			nInterestPoints++;

			// mark location in grid for distance constraint check
			image[offset] = 1;
		}
	}
	#if __TIEMPO__KERNELS__
		hipEventRecord(e_stop);
		hipEventSynchronize(e_stop);
		milliseconds=0;
		hipEventElapsedTime(&milliseconds, e_start, e_stop);
		t_kernel.noParalelo+=milliseconds;
		printf("puntos finales: %f\n",milliseconds);
	#endif

	//hipFree(raw_pointer_cast(d_pCandidateOffsets.data()));
	free(pCandidateOffsets);
	free(nCandidates);
	return nInterestPoints;
}



mutex m_index;
void run(int hilo, char ** fileList, int &count, int &n_veces, int &index, int &contador){

	//int contador=0;
	float milliseconds=0, media=0;
	int ancho,alto;
	bool memoria_reservada=false;
	hipEvent_t start, stop;

	vector<punto> puntosDeInteres(700);

	parametros param;
	bool salir=false;
	CImgDisplay main_disp;
	int j;
	int n_ejec=0;
	tiempos t_kernel;
			// main loop
			//cout << "count: " << count << endl;
			while(contador<n_veces )
			{
				m_index.lock();
				if(index<count){
					j=index;
					index++;
				}
				else {
					j=0;
					index = 0;
					contador++;
				}
				m_index.unlock();

				unsigned char *grayImage = new unsigned char[640*480];

				/**
				 * Convierte la imagen fileList[index] a escala de grises y la guarda
				 * en grayimage
				 */
				ReadBMP(fileList[j], grayImage, ancho, alto);

				if(memoria_reservada==false){
					reserva_memoria(param,ancho,alto);
					memoria_reservada=true;
				}
				CImg<unsigned char> image(grayImage,ancho,alto);
				//main_disp.display(image);


				hipEventCreate(&start);
				hipEventCreate(&stop);
				#if __TIEMPO__TOTAL__
					hipEventRecord(start);
				#endif
				const int nPuntos=cornerDetector(grayImage,puntosDeInteres, ancho, alto, param,t_kernel);
				#if __TIEMPO__TOTAL__
					hipEventRecord(stop);
					hipEventSynchronize(stop);
					milliseconds=0;
					hipEventElapsedTime(&milliseconds, start, stop);
					printf("TIEMPO TOTAL: %f\n",milliseconds);
					media=media+milliseconds;
				#endif

				CImg<unsigned char> img(grayImage,ancho,alto);
				const unsigned char color[3] = {0,100,255};
				for(int i=0;i<nPuntos;i++){
					//img.draw_rectangle(x0,y0,x1,y1,color,1);
					//img.draw_point(puntosDeInteres[i].x,puntosDeInteres[i].y,color);
					img.draw_circle(puntosDeInteres[i].x,puntosDeInteres[i].y,2,color);

				}
				//CImg<unsigned char> image(grayImage,ancho,alto);
				main_disp.display(img);

				free(grayImage);
				n_ejec++;
			}

	main_disp.close();

	cout << "tiempo medio del hilo: " << media/n_ejec << endl;
	cout << "tiempo hilo*n_imagenes/imagenes totales: " << media/((contador)*count) << endl;
	libera_memoria(param);
}



int main(void)

{

	char *fileList[250];
	int count = 0;
	int contador=0;
	GenFileListSorted(SRC_IMAGE, fileList, &count);
	unsigned int n = std::thread::hardware_concurrency();
	n=1;
	thread *v_thread= new thread[n];
	int n_veces=5;
	int index=0;
	for(int t_id=0;t_id<n;t_id++){
		v_thread[t_id]=thread(run,t_id,fileList,std::ref(count),std::ref(n_veces),std::ref(index),std::ref(contador));
	}

	for(int t_id=0;t_id<n;t_id++){
		v_thread[t_id].join();
	}
	delete []v_thread;
}


